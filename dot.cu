/* Includes, system */
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

/* Vector size */
#define N  (4096)

__global__ void d_apply_sigmoid(float *r, int l)
{
	int index = threadIdx.x;
	if (index < l) {
		float val = r[index];
		r[index] = 1.0 / (1.0 + exp(-val));
	}
}

__global__ void d_subs(float *A, float *B, float *C)
{
	A[threadIdx.x] = B[threadIdx.x] - C[threadIdx.x];
}

void d_mmul(hipblasHandle_t &handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}


hipblasStatus_t init_cublas(hipblasHandle_t *handle)
{
	/* Initialize CUBLAS */
	fprintf(stderr, "simpleCUBLAS test running..\n");
	hipblasStatus_t status = hipblasCreate(handle);

	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "Failed to initialize cublas\n");
		fprintf(stderr, "Error number: %d\n", status);
		exit(1);
	}
	return status;
}

/* Main */
int main(int argc, char **argv)
{
    hipblasHandle_t handle;
		init_cublas(&handle);

#if FALSE
    float *h_X, *h_Y, *h_res;
    float *d_X = 0;
    float *d_Y = 0;

    float *d_res = 0;

    int i;


    /* Allocate host memory for the matrices */
    h_X = (float *)malloc(N * sizeof(h_X[0]));
    h_Y = (float *)malloc(N * sizeof(h_Y[0]));
    h_res = (float *)malloc(N * sizeof(h_res[0]));

    /* Fill the matrices with test data */
    for (i = 0; i < N; i++)
    {
        h_X[i] = 2.0;
        h_Y[i] = 2.0;
    }

    /* Allocate device memory for the matrices */
    cudaMalloc((void **)&d_X, N * sizeof(d_X[0]));
    cudaMalloc((void **)&d_Y, N * sizeof(d_Y[0]));
    cudaMalloc((void **)&d_res, N * sizeof(d_res[0]));
    fprintf(stderr, "malloc\n");

    /* Initialize the device matrices with the host matrices */
    cublasSetVector(N, sizeof(h_X[0]), h_X, 1, d_X, 1);
    cublasSetVector(N, sizeof(h_Y[0]), h_Y, 1, d_Y, 1);
		cublasSetVector(N, sizeof(h_res[0]), h_res, 1, d_res, 1);
    fprintf(stderr, "setVector\n");

    /* Performs operation using cublas */
		cublasSdot(handle, N, d_X, 1, d_Y, 1, h_res);
    fprintf(stderr, "sDot\n");

    /* Read the result back */
    //cublasGetVector(N, sizeof(h_res[0]), d_res, 1, h_res, 1);
    //printf("getVector\n");

		int m = 60000;
		int n = (28 * 28 + 1);
#endif

		int X_rows = 60000;
		int X_cols = 784;

		/* actually transposed theta', which makes
		 * setting parameter below easier
		 */
		int theta_rows = X_rows;
		int theta_cols = 1;
		int result_rows = X_rows;
		int result_cols = theta_cols;
		int y_rows = theta_cols;
		int y_cols = theta_rows;

		float *X = (float *)malloc(sizeof(float) * X_rows * X_cols);
		float *d_X;
		float *theta = (float *)malloc(sizeof(float) * theta_rows * theta_cols);
		float *d_theta;
		// Multiplication result
		float *result = (float *)malloc(sizeof(float) * result_rows * result_cols);
		float *d_result;
		// Ys
		float *y = (float *) malloc(sizeof(float) * y_cols);
		float *d_y;

		for (int i = 0; i < X_cols; i++) {
			for (int j = 0; j < X_rows; j++) {
				X[i + j] = i * j;
			}
			y[i] = i;
			theta[i] = 0;
		}

		hipMalloc((void **)&d_X, X_rows * X_cols * sizeof(float));
		hipMalloc((void **)&d_theta, theta_rows * theta_cols * sizeof(float));
		hipMalloc((void **)&d_result, result_rows * result_cols * sizeof(float));
		hipMalloc((void **)&d_y, y_rows * y_cols * sizeof(float));

		/* 
		 * X = {1 1  t = {1
		 *      2 1}      1}
		 */
		hipMemcpy(d_X, X, sizeof(*X) * X_rows * X_cols, hipMemcpyHostToDevice);
		hipMemcpy(d_theta, theta, sizeof(*theta) * theta_rows * theta_cols, hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, sizeof(*y) * y_rows * y_cols, hipMemcpyHostToDevice);

		for (int i = 0; i < 1000; i++) {
			fprintf(stderr, "[%d/%d]\n", i + 1, 100);

			/* Setting these is not trivial:
			 * more info here: http://docs.nvidia.com/cuda/cublas/index.html#cublassetmatrix
			 */
			d_mmul(handle, d_X, d_theta, d_result, X_rows, theta_rows, X_cols);

			d_apply_sigmoid<<<1, result_cols * result_rows>>>(d_result, result_cols * result_rows);

			d_subs<<<1, y_cols>>>(d_result, d_result, d_y);

			float beta = 0;
			float alpha = 1;
								/*handle  transa       transb         m          n         k      alpha   A   lda       B         ldb        beta     C         ldc */ 
			hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, X_cols, result_cols, X_rows, &alpha, d_X, X_cols, d_result,  X_rows,     &beta,  d_result,  X_cols);
		}


		hipMemcpy(result, d_result, sizeof(*result) * result_rows * result_cols, hipMemcpyDeviceToHost);


}
